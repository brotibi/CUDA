
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
 * add.c
 * 
 * Adds two numbers using the CPU.
 */



#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


__global__
void add(int* a, int* b, int* c) {
    *c = *a * *b;
}

int main() {
    int a, b, c;
    int *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, sizeof(int));  
    hipMalloc((void **)&d_b, sizeof(int));
    hipMalloc((void **)&d_c, sizeof(int));
    cudaCheckErrors("hipMalloc fail");

    a = 9;
    b = 10;

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy 1 fail");
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy 2 fail");

    add<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sizeof(int),hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy 3 fail");

    hipDeviceSynchronize();

    printf("GPU says: %d * %d = %d\n", a, b, c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
